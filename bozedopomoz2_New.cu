#define n 100000
#define l 1000

#include <chrono>

#include <hip/hip_runtime.h>

#include <stdlib.h>

#include <stdio.h>

#include <cmath>

#include <cstdio>



#include <cstdlib>

#include <vector>

#include <fstream>

#include <cstring>

#include <algorithm>

//#include <curand.h>

#include <bitset>

#include <cassert>

//kernel accepts a n*l 1d array of bools (bitsetarray), a n*l 1d array of ints(bitsetpairs), where index of pairs are stored,
//a n sized 1d array indexes that keeps track of unused spaces in bitsetpairs and an int pairs that counts the numer of pairs

__global__ void kernel(bool * bitsetarray, int * bitsetpairs, int*indexes, int* pairs) {

  int index = blockIdx.x * blockDim.x + threadIdx.x; // index of a word currently being compared to others
  int number_of_different_bits=0;
  //int bitsetpairs_index=0;
  
  while (index < n) { //repeat the process until we run out of pairs of strings
    for(int j=0;j<n;j++){ //iterate over all n words from bitsetarray
		for(int i=0;i<l;i++){ //iterate over all bits from a given j word
    //TODO: insted of i=0, try i=j, and then dont divide the pairs by 2
		if(bitsetarray[index*l+i]!=bitsetarray[j*l+i]){
		number_of_different_bits++;}
		if(number_of_different_bits>1){ //if number of different bits exceeds 1 we can skip to the next word to save time
		break;}}
		
	
	if(number_of_different_bits==1) //hamming distance equals 1
	{
		(*pairs)++;
	bitsetpairs[index*l+indexes[index]]=j; // save index of word <j> at row <index> in a free column
	indexes[index]++; //move to the next free place
	}	
	number_of_different_bits=0;
        }

    
  

    index += blockDim.x * gridDim.x; //move to the next word to be compared to others
  }

}

void load_from_file(char * file_name, bool * bitsetarray) {
  FILE * a_file = fopen(file_name, "r");

  if (a_file == NULL) {
    printf("unable to load file\n");
    return;
  }

  char temp;

  for (int i = 0; i < n; i++) {
      for(int j=0;j<l;j++){
      if (!fscanf(a_file, "%c\n", & temp)) {
        printf("File loading error!\n");
        return;
      }

      if (temp == '1') {
        bitsetarray[i*l+j]=true;
      } 
      else
      bitsetarray[i*l+j]=false;
      }
  }
}
void print_word(bool*bitsetarray, int index){
	for(int i=0;i<l;i++){
		printf("%d",bitsetarray[index*l+i]);	
	}
	printf("\n");
	
}
void print_solution(bool* bitsetarray,int*bitsetpairs, int* indexes, int *pairs){
printf("Number of  pairs found = %d\n", (*pairs)/2); //we divide by two, cause every pair as of now is counted twice
for(int i=0;i<n;i++){
  printf("Words with Hamming distance equal to 1 with word: ");
  print_word(bitsetarray,i);
  printf("are:\n");
  for(int j=0;j<indexes[i];j++){
      print_word(bitsetarray,bitsetpairs[i*l+j]);

  }
}

}
//just prints the contents of bitsetpairs
void test_bitsetpairs(int *bitsetpairs){
	
	for(int i=0;i<n;i++)
	{
		for(int j=0;j<l;j++){
			printf("%d",bitsetpairs[i*l+j]);
		}
		printf("\n");
	}
	
	
}

int main(int argc, char ** argv) {

  if (argc < 4) {
    printf("threads_per_block max_blocks file_with_data\n");
    return 1;
  }

  //the below are of no use currently
  int dev = 0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties( & deviceProp, dev);

  unsigned int maxThreads = deviceProp.maxThreadsPerBlock;

  //event creation
  hipEvent_t start, stop;
  hipEventCreate( & start);
  hipEventCreate( & stop);
  hipEvent_t startbfs, stopbfs;
  hipEventCreate( & startbfs);
  hipEventCreate( & stopbfs);

  hipEvent_t start_memalloc, stop_memalloc;
  hipEventCreate( & start_memalloc);
  hipEventCreate( & stop_memalloc);

  hipEvent_t start_reading, stop_reading;
  hipEventCreate( & start_reading);
  hipEventCreate( & stop_reading);

  hipEvent_t start_copying, stop_copying;
  hipEventCreate( & start_copying);
  hipEventCreate( & stop_copying);

  auto begin = std::chrono::high_resolution_clock::now();

  //read arguments
  const unsigned int threadsPerBlock = atoi(argv[1]);
  const unsigned int maxBlocks = atoi(argv[2]);
  char * file_name = argv[3];

  //auto bitsetarray = new bitset<l>[n];
  bool *bitsetarray = new bool[n*l];
  //std::bitset<l> bitsetarray[n]; 

  //read data
  hipEventRecord(start_reading);
  load_from_file(file_name, bitsetarray);
  hipEventRecord(stop_reading);


  //initialization of variables
  //auto bitsetpairs = new bitset<l>[n];
  int * bitsetpairs=new int[n*l];
  int * indexes=new int[n];
  int *pairs=new int;
  (*pairs)=0;
  memset(bitsetpairs,0,n*l*sizeof(int));
  memset(indexes,0,n*sizeof(int));
  int * indexes_dev=new int[n];
  int * bitsetpairs_dev=new int[n*l];
  int *pairs_dev=new int;
  bool * bitsetarray_dev=new bool[n*l];
  //std::vector <int> *bitsetpairs=new std::vector<int>[n];
  //std::vector<int> *bitsetpairs_dev;// = new vector<int>[n];
  //std::bitset<l> *bitsetarray_dev;// =new bitset<l>[n];

  hipEventRecord(start_memalloc);
  hipMalloc( & pairs_dev,  sizeof(int));
  hipMalloc( & indexes_dev, n * sizeof(int));
  hipMalloc( & bitsetpairs_dev, n *l* sizeof(int));
  hipMalloc( & bitsetarray_dev, n *l* sizeof(bool));
  hipEventRecord(stop_memalloc);


  hipEventRecord(start_copying);
  hipMemcpy(pairs_dev, pairs, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(bitsetarray_dev, bitsetarray, n *l* sizeof(bool), hipMemcpyHostToDevice);
  hipMemcpy(indexes_dev, indexes, n * sizeof(int), hipMemcpyHostToDevice);
  hipEventRecord(stop_copying);
  
  hipEventRecord(start);
  kernel <<< maxBlocks, threadsPerBlock >>> (bitsetarray_dev,bitsetpairs_dev,indexes_dev,pairs_dev);
    hipEventRecord(stop);

  hipMemcpy(bitsetpairs, bitsetpairs_dev, n *l* sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(indexes, indexes_dev, n * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(pairs, pairs_dev, sizeof(int), hipMemcpyDeviceToHost);
  
  //test_bitsetpairs(bitsetpairs);
  

  //pairs are actually counted based on values in "indexes" since it keeps track of number of pairs for each word
  //so the pair counting in kernel can be removed, Im just scared to modify anything without testing it
  (*pairs)=0;
  for(int i=0;i<n;i++)
  (*pairs)+=indexes[i];
  


  //the function print_solution also prints pairs for each word, in the case when n and l are big its output is rather unreadable
  //print_solution(bitsetarray,bitsetpairs,indexes,pairs);
	printf("Number of pairs found: %d\n",(*pairs)/2);


  delete[] bitsetarray;
  delete[] bitsetpairs;
  hipFree(bitsetarray_dev);
  hipFree(bitsetpairs_dev);
  //untested changes:
  delete[] indexes;
  delete pairs;
  hipFree(indexes_dev);
  hipFree(pairs_dev);
  //

  hipEventSynchronize(stop);


  //print results of time measurments
  float millisecondscopying = 0;
  hipEventElapsedTime( & millisecondscopying, start_copying, stop_copying);
  printf("Data Copying: %.3f seconds.\n", 0.001 * millisecondscopying);
  float millisecondsreading = 0;
  hipEventElapsedTime( & millisecondsreading, start_reading, stop_reading);
  printf("Data Loading: %.3f seconds.\n", 0.001 * millisecondsreading);
  float milliseconds = 0;
  hipEventElapsedTime( & milliseconds, start, stop);
  printf("Kernel: %.3f seconds.\n", 0.001 * milliseconds);
  float millisecondsmem = 0;
  hipEventElapsedTime( & millisecondsmem, start_memalloc, stop_memalloc);
  printf("MemAlloc: %.3f seconds.\n", 0.001 * millisecondsmem);
  auto end = std::chrono::high_resolution_clock::now();
  auto elapsed = std::chrono::duration_cast < std::chrono::nanoseconds > (end - begin);

  printf("Time measured(total time): %.3f seconds.\n", elapsed.count() * 1e-9);
  return 0;

}
